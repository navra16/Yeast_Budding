#include "System.h"
#include "Nodes2Triangles.h"
#include "SystemStructures.h"



//WARNING: function must not reset coordInfoVecs.nodeForceX etc. 
void ComputeNodes2Triangles(
    CoordInfoVecs& coordInfoVecs,
    GeneralParams& generalParams,
    AuxVecs& auxVecs) {    
    
    /*thrust::fill(coordInfoVecs.tempNodeForceXReduced.begin(),coordInfoVecs.tempNodeForceXReduced.end(),0.0);
    thrust::fill(coordInfoVecs.tempNodeForceYReduced.begin(),coordInfoVecs.tempNodeForceYReduced.end(),0.0);
    thrust::fill(coordInfoVecs.tempNodeForceZReduced.begin(),coordInfoVecs.tempNodeForceZReduced.end(),0.0);
    thrust::fill(coordInfoVecs.tempNodeForceXUnreduced.begin(),coordInfoVecs.tempNodeForceXUnreduced.end(),0.0);
    thrust::fill(coordInfoVecs.tempNodeForceYUnreduced.begin(),coordInfoVecs.tempNodeForceYUnreduced.end(),0.0);
    thrust::fill(coordInfoVecs.tempNodeForceZUnreduced.begin(),coordInfoVecs.tempNodeForceZUnreduced.end(),0.0);*/

    //CVec4 init(0.0, 0.0, 0.0, 0.0); 
    thrust::counting_iterator<int> begin(0);

    thrust::transform(  
        thrust::make_zip_iterator(
            thrust::make_tuple(
                begin,
                auxVecs.id_bucket.begin()
            )),
        
        thrust::make_zip_iterator(
            thrust::make_tuple(
                begin,
                auxVecs.id_bucket.begin())) + generalParams.maxNodeCount,

        thrust::make_zip_iterator(
            thrust::make_tuple(
                coordInfoVecs.nodes2Triangles_1.begin(),
                coordInfoVecs.nodes2Triangles_2.begin(),
                coordInfoVecs.nodes2Triangles_3.begin(),
                coordInfoVecs.nodes2Triangles_4.begin(),
                coordInfoVecs.nodes2Triangles_5.begin(),
                coordInfoVecs.nodes2Triangles_6.begin(),
                coordInfoVecs.nodes2Triangles_7.begin(),
                coordInfoVecs.nodes2Triangles_8.begin(),
                coordInfoVecs.nodes2Triangles_9.begin()
               )),

        Nodes2TrianglesFunctor(
            
            coordInfoVecs.num_triangles,

            thrust::raw_pointer_cast(coordInfoVecs.triangles2Nodes_1.data()),
            thrust::raw_pointer_cast(coordInfoVecs.triangles2Nodes_2.data()),
            thrust::raw_pointer_cast(coordInfoVecs.triangles2Nodes_3.data()),              
                          
                thrust::raw_pointer_cast(auxVecs.id_value_expanded.data()),
                thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
                thrust::raw_pointer_cast(auxVecs.keyEnd.data()))
            );
                     
};

